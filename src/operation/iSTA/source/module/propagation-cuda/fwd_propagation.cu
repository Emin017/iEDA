#include "hip/hip_runtime.h"
// ***************************************************************************************
// Copyright (c) 2023-2025 Peng Cheng Laboratory
// Copyright (c) 2023-2025 Institute of Computing Technology, Chinese Academy of
// Sciences Copyright (c) 2023-2025 Beijing Institute of Open Source Chip
//
// iEDA is licensed under Mulan PSL v2.
// You can use this software according to the terms and conditions of the Mulan
// PSL v2. You may obtain a copy of Mulan PSL v2 at:
// http://license.coscl.org.cn/MulanPSL2
//
// THIS SOFTWARE IS PROVIDED ON AN "AS IS" BASIS, WITHOUT WARRANTIES OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO
// NON-INFRINGEMENT, MERCHANTABILITY OR FIT FOR A PARTICULAR PURPOSE.
//
// See the Mulan PSL v2 for more details.
// ***************************************************************************************
/**
 * @file fwd_propagation.cu
 * @author simin tao (taosm@pcl.ac.cn)
 * @brief The fwd propagation using GPU.
 * @version 0.1
 * @date 2025-01-15
 *
 */
#include <hip/hip_runtime.h>

#include "fwd_propagation.cuh"
// #include "sta/StaGraph.hh"

namespace ista {

/**
 * @brief device function for lut delay arc, using slew and load
 *
 * @param trans_type
 * @param in_slew input slew
 * @param out_load output load
 * @param snk_slew the lut snk vertex slew for store value.
 * @param arc_delay the arc delay for store value
 * @return __device__
 */
__device__ void lut_inst_slew_delay(GPU_Vertex_Data* in_slew,
                                    GPU_Vertex_Data* out_load,
                                    GPU_Vertex_Data* snk_slew,
                                    GPU_Vertex_Data* arc_delay) {
  // TODO(to taosimin), call gpu lut table.
  // store the lut value
}

/**
 * @brief device function for lut check arc, using slew and snk slew.
 *
 * @param in_slew
 * @param snk_slew
 * @param arc_delay
 * @return __device__
 */
__device__ void lut_constraint_delay(GPU_Vertex_Data* in_slew,
                                     GPU_Vertex_Data* snk_slew,
                                     GPU_Vertex_Data* arc_delay) {
  // TODO(to taosimin), call gpu lut table.
  // store the lut value
  GPU_Fwd_Data one_src_slew_data;
  FOREACH_GPU_FWD_DATA((*in_slew), one_src_slew_data) {}
}

/**
 * @brief device function for lut net arc, using input slew and node table.
 *
 * @param in_slew
 * @param delay_data
 * @param impulse_data
 * @return __device__
 */
__device__ void lut_net_slew_delay(GPU_Vertex_Data* in_slew,
                                   GPU_Vertex_Data* delay_data,
                                   GPU_Vertex_Data* impulse_data,
                                   GPU_Vertex_Data* snk_slew,
                                   GPU_Vertex_Data* arc_delay) {}

/**
 * @brief propagate the bfs arcs in cuda kernal.
 *
 * @param the_graph
 * @param propagated_arcs
 * @return __global__
 */
__global__ void propagate_fwd(GPU_Graph the_graph,
                              GPU_BFS_Propagated_Arc propagated_arcs) {
  // current thread id
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < propagated_arcs._num_arcs) {
    unsigned current_arc_id = propagated_arcs._arc_start_addr[i];
    GPU_Arc current_arc = the_graph._arcs[current_arc_id];
    GPU_Arc_Type current_arc_type = current_arc._arc_type;
    unsigned src_vertex_id = current_arc._src_vertex_id;
    unsigned snk_vertex_id = current_arc._snk_vertex_id;
    auto src_vertex = the_graph._vertices[src_vertex_id];
    auto snk_vertex = the_graph._vertices[snk_vertex_id];

    if (current_arc_type == kInstDelayArc) {
      // lut table for snk arc slew and arc delay use src slew and out cap.
      lut_inst_slew_delay(&src_vertex._slew_data, &snk_vertex._node_cap_data,
                          &snk_vertex._slew_data, &current_arc._delay_values);
    } else if (current_arc_type == kInstCheckArc) {
      // lut table for get constrain value for check arc.
      lut_constraint_delay(&src_vertex._slew_data, &snk_vertex._slew_data,
                           &current_arc._delay_values);
    } else {
      // for net arc
      // lut net output slew and delay.
      lut_net_slew_delay(&src_vertex._slew_data, &snk_vertex._node_delay_data,
                         &snk_vertex._node_impulse_data, &snk_vertex._slew_data,
                         &current_arc._delay_values);
    }
  }
}
#if 0
/**
 * @brief copy sta graph to gpu sta graph.
 *
 */
GPU_Graph build_gpu_sta_graph(StaGraph* the_cpu_graph) {
  GPU_Graph the_gpu_graph;
  return the_gpu_graph;
}

/**
 * @brief copyback gpu data to cpu sta graph.
 *
 * @param the_cpu_graph
 * @param the_gpu_graph
 */
void update_sta_graph(StaGraph* the_cpu_graph, GPU_Graph the_gpu_graph) {}

/**
 * @brief The interface function for the fwd function.
 * first, build gpu graph
 * for vertex, copy slew data, load, at data, node delay, node impulse
 * for arc, set src and snk id
 * then, propagate level by level.
 */
void gpu_propagate_fwd(
    StaGraph* the_cpu_graph,
    std::map<unsigned, std::vector<StaArc*>>& level_to_arcs) {
  auto the_gpu_graph = build_gpu_sta_graph(the_cpu_graph);
  GPU_BFS_Propagated_Arc propagate_arcs;
  //TODO(to taosimin), copy arc id to gpu bfs propagated arc.
  for (auto& [level, the_arcs] : level_to_arcs) {
    propagate_fwd<<<1, 1000>>>(the_gpu_graph, propagate_arcs);
  }

  update_sta_graph(the_cpu_graph, the_gpu_graph);
}
#endif

}  // namespace ista