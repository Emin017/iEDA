#include "hip/hip_runtime.h"
// ***************************************************************************************
// Copyright (c) 2023-2025 Peng Cheng Laboratory
// Copyright (c) 2023-2025 Institute of Computing Technology, Chinese Academy of
// Sciences Copyright (c) 2023-2025 Beijing Institute of Open Source Chip
//
// iEDA is licensed under Mulan PSL v2.
// You can use this software according to the terms and conditions of the Mulan
// PSL v2. You may obtain a copy of Mulan PSL v2 at:
// http://license.coscl.org.cn/MulanPSL2
//
// THIS SOFTWARE IS PROVIDED ON AN "AS IS" BASIS, WITHOUT WARRANTIES OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO
// NON-INFRINGEMENT, MERCHANTABILITY OR FIT FOR A PARTICULAR PURPOSE.
//
// See the Mulan PSL v2 for more details.
// ***************************************************************************************
/**
 * @file fwd_propagation.cu
 * @author simin tao (taosm@pcl.ac.cn)
 * @brief The fwd propagation using GPU.
 * @version 0.1
 * @date 2025-01-15
 *
 */
#include <hip/hip_runtime.h>

#include <map>

#include "fwd_propagation.cuh"
#include "gpu/cuda_common.cuh"
#include "propagation.cuh"

namespace ista {

/**
 * @brief device function for lut delay arc, using slew and load
 *
 * @param trans_type
 * @param in_slew input slew
 * @param out_load output load
 * @param snk_slew the lut snk vertex slew for store value.
 * @param arc_delay the arc delay for store value
 * @return __device__
 */
__device__ void lut_inst_slew_delay(GPU_Vertex_Data* in_slew,
                                    GPU_Vertex_Data* out_load,
                                    GPU_Vertex_Data* snk_slew,
                                    GPU_Vertex_Data* arc_delay) {
  // TODO(to taosimin), call gpu lut table.
  // store the lut value
}

/**
 * @brief device function for lut check arc, using slew and snk slew.
 *
 * @param in_slew
 * @param snk_slew
 * @param arc_delay
 * @return __device__
 */
__device__ void lut_constraint_delay(GPU_Graph* the_graph,
                                     GPU_Vertex_Data* in_slew,
                                     GPU_Vertex_Data* snk_slew,
                                     GPU_Vertex_Data* arc_delay) {
  // TODO(to taosimin), call gpu lut table.
  // store the lut value
  GPU_Fwd_Data one_src_slew_data;
  FOREACH_GPU_FWD_DATA(the_graph->_flatten_slew_data, (*in_slew),
                       one_src_slew_data) {}
}

/**
 * @brief device function for lut net arc, using input slew and node table.
 *
 * @param in_slew
 * @param delay_data
 * @param impulse_data
 * @return __device__
 */
__device__ void lut_net_slew_delay(GPU_Vertex_Data* in_slew,
                                   GPU_Vertex_Data* delay_data,
                                   GPU_Vertex_Data* impulse_data,
                                   GPU_Vertex_Data* snk_slew,
                                   GPU_Vertex_Data* arc_delay) {}

/**
 * @brief propagate the bfs arcs in cuda kernal.
 *
 * @param the_graph
 * @param propagated_arcs
 * @return __global__
 */
__global__ void propagate_fwd(GPU_Graph the_graph,
                              GPU_BFS_Propagated_Arc propagated_arcs) {
  // current thread id
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < propagated_arcs._num_arcs) {
    unsigned current_arc_id = propagated_arcs._arc_index[i];
    GPU_Arc current_arc = the_graph._arcs[current_arc_id];
    GPU_Arc_Type current_arc_type = current_arc._arc_type;
    unsigned src_vertex_id = current_arc._src_vertex_id;
    unsigned snk_vertex_id = current_arc._snk_vertex_id;
    auto src_vertex = the_graph._vertices[src_vertex_id];
    auto snk_vertex = the_graph._vertices[snk_vertex_id];

    if (current_arc_type == kInstDelayArc) {
      // lut table for snk arc slew and arc delay use src slew and out cap.
      lut_inst_slew_delay(&src_vertex._slew_data, &snk_vertex._node_cap_data,
                          &snk_vertex._slew_data, &current_arc._delay_values);
    } else if (current_arc_type == kInstCheckArc) {
      // lut table for get constrain value for check arc.
      lut_constraint_delay(&the_graph, &src_vertex._slew_data,
                           &snk_vertex._slew_data, &current_arc._delay_values);
    } else {
      // for net arc
      // lut net output slew and delay.
      lut_net_slew_delay(&src_vertex._slew_data, &snk_vertex._node_delay_data,
                         &snk_vertex._node_impulse_data, &snk_vertex._slew_data,
                         &current_arc._delay_values);
    }
  }
}

/**
 * @brief copy sta graph to gpu sta graph.
 *
 */
GPU_Graph copy_from_sta_graph(GPU_Graph& the_cpu_graph,
                              unsigned vertex_data_size,
                              unsigned arc_data_size) {
  const unsigned num_stream = 7;
  hipStream_t stream[num_stream];
  for (unsigned index = 0; index < num_stream; ++index) {
    hipStreamCreate(&stream[index]);
  }

  GPU_Graph the_gpu_graph;

  CUDA_CHECK(hipMallocAsync((void**)&the_gpu_graph._vertices,
                             the_gpu_graph._num_vertices * sizeof(GPU_Vertex),
                             stream[0]));

  CUDA_CHECK(hipMallocAsync((void**)&the_gpu_graph._arcs,
                             the_gpu_graph._num_arcs * sizeof(GPU_Arc),
                             stream[1]));

  CUDA_CHECK(hipMallocAsync((void**)&the_gpu_graph._flatten_slew_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             stream[2]));

  CUDA_CHECK(hipMallocAsync((void**)&the_gpu_graph._flatten_at_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             stream[3]));

  CUDA_CHECK(hipMallocAsync((void**)&the_gpu_graph._flatten_node_cap_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             stream[4]));

  CUDA_CHECK(hipMallocAsync((void**)&the_gpu_graph._flatten_node_impulse_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             stream[5]));

  CUDA_CHECK(hipMallocAsync((void**)&the_gpu_graph._flatten_arc_delay_data,
                             arc_data_size * sizeof(GPU_Fwd_Data), stream[6]));

  for (unsigned index = 0; index < num_stream; ++index) {
    hipStreamSynchronize(stream[index]);
  }

  CUDA_CHECK(hipMemcpyAsync(the_gpu_graph._vertices, the_cpu_graph._vertices,
                             the_gpu_graph._num_vertices * sizeof(GPU_Vertex),
                             hipMemcpyHostToDevice, stream[0]));

  CUDA_CHECK(hipMemcpyAsync(the_gpu_graph._arcs, the_cpu_graph._arcs,
                             the_gpu_graph._num_arcs * sizeof(GPU_Arc),
                             hipMemcpyHostToDevice, stream[1]));

  CUDA_CHECK(hipMemcpyAsync(the_gpu_graph._flatten_slew_data,
                             the_cpu_graph._flatten_slew_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[2]));

  CUDA_CHECK(hipMemcpyAsync(the_gpu_graph._flatten_at_data,
                             the_cpu_graph._flatten_at_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[3]));

  CUDA_CHECK(hipMemcpyAsync(the_gpu_graph._flatten_node_cap_data,
                             the_cpu_graph._flatten_node_cap_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[4]));

  CUDA_CHECK(hipMemcpyAsync(the_gpu_graph._flatten_node_impulse_data,
                             the_cpu_graph._flatten_node_impulse_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[5]));

  CUDA_CHECK(hipMemcpyAsync(the_gpu_graph._flatten_arc_delay_data,
                             the_cpu_graph._flatten_arc_delay_data,
                             arc_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[6]));

  for (unsigned index = 0; index < num_stream; ++index) {
    hipStreamSynchronize(stream[index]);
  }

  for (unsigned index = 0; index < num_stream; ++index) {
    hipStreamDestroy(stream[index]);
  }
  return the_gpu_graph;
}

/**
 * @brief copyback gpu data to cpu sta graph.
 *
 * @param the_cpu_graph
 * @param the_gpu_graph
 */
void copy_to_sta_graph(GPU_Graph& the_cpu_graph, GPU_Graph& the_gpu_graph,
                       unsigned vertex_data_size, unsigned arc_data_size) {
  const unsigned num_stream = 5;
  hipStream_t stream[num_stream];
  for (unsigned index = 0; index < num_stream; ++index) {
    hipStreamCreate(&stream[index]);
  }

  CUDA_CHECK(hipMemcpyAsync(the_cpu_graph._flatten_slew_data,
                             the_gpu_graph._flatten_slew_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[0]));

  CUDA_CHECK(hipMemcpyAsync(the_cpu_graph._flatten_at_data,
                             the_gpu_graph._flatten_at_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[1]));

  CUDA_CHECK(hipMemcpyAsync(the_cpu_graph._flatten_node_cap_data,
                             the_gpu_graph._flatten_node_cap_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[2]));

  CUDA_CHECK(hipMemcpyAsync(the_cpu_graph._flatten_node_impulse_data,
                             the_gpu_graph._flatten_node_impulse_data,
                             vertex_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[3]));

  CUDA_CHECK(hipMemcpyAsync(the_cpu_graph._flatten_arc_delay_data,
                             the_gpu_graph._flatten_arc_delay_data,
                             arc_data_size * sizeof(GPU_Fwd_Data),
                             hipMemcpyHostToDevice, stream[4]));

  for (unsigned index = 0; index < num_stream; ++index) {
    hipStreamSynchronize(stream[index]);
  }

  for (unsigned index = 0; index < num_stream; ++index) {
    hipStreamDestroy(stream[index]);
  }
}

/**
 * @brief The interface function for the fwd function.
 * first, build gpu graph
 * for vertex, copy slew data, load, at data, node delay, node impulse
 * for arc, set src and snk id
 * then, propagate level by level.
 */
void gpu_propagate_fwd(
    GPU_Graph& the_cpu_graph, unsigned vertex_data_size, unsigned arc_data_size, 
    std::map<unsigned, GPU_BFS_Propagated_Arc>& level_to_arcs, Lib_Data_GPU& lib_data) {
  auto the_gpu_graph =
      copy_from_sta_graph(the_cpu_graph, vertex_data_size, arc_data_size);

  // TODO(to taosimin), copy arc id to gpu bfs propagated arc.
  for (auto& [level, the_arcs] : level_to_arcs) {
    propagate_fwd<<<1, 1000>>>(the_gpu_graph, the_arcs);
  }

  copy_to_sta_graph(the_cpu_graph, the_gpu_graph, vertex_data_size, arc_data_size);
}

}  // namespace ista